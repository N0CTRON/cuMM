#include <hip/hip_runtime.h>


#include <thread>

template <typename devType>
class cuMM
{
public:
	devType* data;

	void malloc(size_t elems) { hipMalloc(&data, sizeof(devType) * elems); }
	void free()
	{
		if (data != nullptr) //<-- Branching neccessary :(
		{
			hipFree(data);
			data = nullptr;
		}
	}

	void resize(size_t newSize)
	{
		return; //Next commit
	};

	cuMM() {};
	cuMM(size_t elems) { hipMalloc(&data, sizeof(devType) * elems); }; //same as cuMalloc cuz overhead 'n stuff
	~cuMM() { std::thread(free); }
};