#include <hip/hip_runtime.h>


#include <thread>

template <typename devType>
class cuMM
{
public:
	devType* data = nullptr;
	size_t currentSizeBytes = NULL;

	void malloc(size_t elems)
	{
		currentSizeBytes = sizeof(devType) * elems;
		hipMalloc(&data, currentSizeBytes);
	}

	void free()
	{
		if (data != nullptr) //<-- Branching neccessary :(
		{
			hipFree(data);
			data = nullptr;
		}
	}

	void resize(size_t newElems)
	{
		if (newElems == 0)
		{
			free();
			return;
		}

		devType* newPtr = nullptr;
		size_t newSize = sizeof(devType) * newElems;
		hipMalloc(&newPtr, newSize);

		if (newPtr != nullptr)
		{
			hipMemcpy(newPtr, data, currentSizeBytes < newSize ? currentSizeBytes : newSize, hipMemcpyDeviceToDevice);
			hipFree(data);
			data = newPtr;
			currentSizeBytes = newSize;
		}
		else throw "CuMM couldn't resize: \"newPtr\" is nullPtr. Keeping old memory.\n";
	}

	size_t size() { return currentSizeBytes / sizeof(devType); }
	size_t sizeBytes() { return currentSizeBytes; }

	cuMM() {};
	cuMM(size_t elems) //same as cuMalloc cuz overhead 'n stuff
	{
		currentSizeBytes = sizeof(devType) * elems;
		hipMalloc(&data, currentSizeBytes);
	};
	~cuMM() { std::thread(free); }

	devType& operator[] (const size_t& index) { return this->data[index]; }
};
