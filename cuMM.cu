#include <hip/hip_runtime.h>


#include <thread>

template <typename devType>
class cuMM
{
public:
	devType* data = nullptr;
	std::uint64_t currentSizeBytes = NULL;

	void malloc(std::uint64_t elems)
	{
		currentSizeBytes = sizeof(devType) * elems;
		hipMalloc(&data, currentSizeBytes);
	}

	void free()
	{
		if (data != nullptr) //<-- Branching neccessary :(
		{
			hipFree(data);
			data = nullptr;
		}
	}

	void resize(std::uint64_t newElems)
	{
		if (newElems == 0)
		{
			free();
			return;
		}

		devType* newPtr = nullptr;
		std::uint64_t newSize = sizeof(devType) * newElems;
		hipMalloc(&newPtr, newSize);

		if (newPtr != nullptr)
		{
			hipMemcpy(newPtr, data, currentSizeBytes < newSize ? currentSizeBytes : newSize, hipMemcpyDeviceToDevice);
			hipFree(data);
			data = newPtr;
			currentSizeBytes = newSize;
		}
		else throw "CuMM couldn't resize: \"newPtr\" is nullPtr. Keeping old memory.\n";
	}

	template <typename devArray>
	void copy(devArray toCopy, std::uint64_t size, bool toOrFrom) //true = to / to CUDA, false = from / to host
	{
		if (toOrFrom && data != nullptr) hipMemcpy(data, (void*)toCopy, std::min(currentSizeBytes, size), hipMemcpyHostToDevice);
		else if (!toOrFrom && data != nullptr) hipMemcpy((void*)toCopy, data, std::min(currentSizeBytes, size), hipMemcpyDeviceToHost);
	}

	std::uint64_t size() { return currentSizeBytes / sizeof(devType); }
	std::uint64_t sizeBytes() { return currentSizeBytes; }

	cuMM() {};
	cuMM(std::uint64_t elems) //same as cuMalloc cuz overhead 'n stuff
	{
		currentSizeBytes = sizeof(devType) * elems;
		hipMalloc(&data, currentSizeBytes);
	};
	~cuMM() { std::thread(free); }

	devType& operator[] (const std::uint64_t& index) { return this->data[index]; }
};
