#include <hip/hip_runtime.h>


#include <thread>

template <typename devType>
class cuMM
{
public:
	devType* data = nullptr;
	size_t currentSizeBytes = NULL;

	void malloc(size_t elems)
	{
		currentSizeBytes = sizeof(devType) * elems;
		hipMalloc(&data, currentSizeBytes);
	}

	void free()
	{
		if (data != nullptr) //<-- Branching neccessary :(
		{
			hipFree(data);
			data = nullptr;
		}
	}

	void resize(size_t newSize)
	{
		devType* newPtr;
		hipMalloc(&newPtr, newSize);
		if (newPtr != nullptr)
		{
			hipMemcpy(newPtr, data, currentSizeBytes < newSize ? currentSizeBytes : newSize, hipMemcpyHostToHost);
			hipFree(data);
		}
		data = newPtr;
	};

	cuMM() {};

	cuMM(size_t elems) //same as cuMalloc cuz overhead 'n stuff
	{
		currentSizeBytes = sizeof(devType) * elems;
		hipMalloc(&data, currentSizeBytes);
	};

	~cuMM() { std::thread(free); }
};
